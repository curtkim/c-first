#include <stdio.h>
#include <hipcub/hipcub.hpp>

void scan_on_device() {
  // Declare, allocate, and initialize device pointers for input and output
  int num_items = 7;
  int *d_in;
  int h_in[] = {8, 6, 7, 5, 3, 0, 9};
  int sz = sizeof(h_in) / sizeof(h_in[0]);
  int *d_out; // e.g., [ , , , , , , ]
  hipMalloc(&d_in, sz * sizeof(h_in[0]));
  hipMalloc(&d_out, sz * sizeof(h_in[0]));
  hipMemcpy(d_in, h_in, sz * sizeof(h_in[0]), hipMemcpyHostToDevice);

  printf("\nInput:\n");
  for (int i = 0; i < sz; i++)
    printf("%d ", h_in[i]);

  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);

  // Run inclusive prefix sum
  hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items);
  // d_out s<-- [8, 14, 21, 26, 29, 29, 38]
  hipMemcpy(h_in, d_out, sz * sizeof(h_in[0]), hipMemcpyDeviceToHost);

  printf("\nOutput:\n");
  for (int i = 0; i < sz; i++)
    printf("%d ", h_in[i]);
  printf("\n");
}

int main(void) {
  scan_on_device();
  return 0;
}
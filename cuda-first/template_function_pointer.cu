// from https://leimao.github.io/blog/Pass-Function-Pointers-to-Kernels-CUDA/

#include <hip/hip_runtime.h>
#include <iostream>

// Since C++ 11
template<typename T>
using func_t = T (*) (T, T);

template <typename T>
__device__ T add_func (T x, T y)
{
return x + y;
}

template <typename T>
__device__ T mul_func (T x, T y)
{
return x * y;
}

// Required for functional pointer argument in kernel function
// Static pointers to device functions
template <typename T>
__device__ func_t<T> p_add_func = add_func<T>;
template <typename T>
__device__ func_t<T> p_mul_func = mul_func<T>;


template <typename T>
__global__ void kernel(func_t<T> op, T * d_x, T * d_y, T * result)
{
  *result = (*op)(*d_x, *d_y);
}

template <typename T>
void test(T x, T y)
{
  func_t<T> h_add_func;
  func_t<T> h_mul_func;

  T * d_x, * d_y;
  hipMalloc(&d_x, sizeof(T));
  hipMalloc(&d_y, sizeof(T));
  hipMemcpy(d_x, &x, sizeof(T), hipMemcpyHostToDevice);
  hipMemcpy(d_y, &y, sizeof(T), hipMemcpyHostToDevice);

  T result;
  T * d_result, * h_result;
  hipMalloc(&d_result, sizeof(T));
  h_result = &result;

  // Copy device function pointer to host side
  hipMemcpyFromSymbol(&h_add_func, HIP_SYMBOL(p_add_func<T>), sizeof(func_t<T>));
  hipMemcpyFromSymbol(&h_mul_func, HIP_SYMBOL(p_mul_func<T>), sizeof(func_t<T>));

  kernel<T><<<1,1>>>(h_add_func, d_x, d_y, d_result);
  hipDeviceSynchronize();
  hipMemcpy(h_result, d_result, sizeof(T), hipMemcpyDeviceToHost);
  std::cout << "Sum: " << result << std::endl;

  kernel<T><<<1,1>>>(h_mul_func, d_x, d_y, d_result);
  hipDeviceSynchronize();
  hipMemcpy(h_result, d_result, sizeof(T), hipMemcpyDeviceToHost);
  std::cout << "Product: " << result << std::endl;
}

int main()
{
  std::cout << "Test int for type int ..." << std::endl;
  test<int>(2.05, 10.00);

  std::cout << "Test float for type float ..." << std::endl;
  test<float>(2.05, 10.00);

  std::cout << "Test double for type double ..." << std::endl;
  test<double>(2.05, 10.00);
}
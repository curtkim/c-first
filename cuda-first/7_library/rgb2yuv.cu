
#include <hip/hip_runtime.h>
// from https://www.programmersought.com/article/58352347721/
// super-clean formula
#define RGB2Y(R, G, B)  ( 16  + 0.183f * (R) + 0.614f * (G) + 0.062f * (B) )
#define RGB2U(R, G, B)  ( 128 - 0.101f * (R) - 0.339f * (G) + 0.439f * (B) )
#define RGB2V(R, G, B)  ( 128 + 0.439f * (R) - 0.399f * (G) - 0.040f * (B) )

#define YUV2R(Y, U, V) ( 1.164f *((Y) - 16) + 1.792f * ((V) - 128) )
#define YUV2G(Y, U, V) ( 1.164f *((Y) - 16) - 0.213f *((U) - 128) - 0.534f *((V) - 128) )
#define YUV2B(Y, U, V) ( 1.164f *((Y) - 16) + 2.114f *((U) - 128))

#define CLIPVALUE(x, minValue, maxValue) ((x) < (minValue) ? (minValue) : ((x) > (maxValue) ? (maxValue) : (x)))

__global__ static void __RgbToYuv420p(const unsigned char* dpRgbData, size_t rgbPitch, unsigned char* dpYuv420pData, size_t yuv420Pitch, int width, int height)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int w = index % yuv420Pitch;
  int h = index / yuv420Pitch;

  if (w >= width || h >= height)
    return;

  unsigned char* dp_y_data = dpYuv420pData;
  unsigned char* dp_u_data = dp_y_data + height * yuv420Pitch;
  unsigned char* dp_v_data = dp_u_data + height * yuv420Pitch / 4;

  unsigned char r = dpRgbData[h * rgbPitch + w * 3 + 0];
  unsigned char g = dpRgbData[h * rgbPitch + w * 3 + 1];
  unsigned char b = dpRgbData[h * rgbPitch + w * 3 + 2];

  dp_y_data[h   * yuv420Pitch + w] = (unsigned char)(CLIPVALUE(RGB2Y(r, g, b), 0, 255));
  int num = h / 2 * width / 2 + w / 2;
  int offset = num / width * (yuv420Pitch - width);

  if (h % 2 == 0 && w % 2 == 0)
  {
    dp_u_data[num + offset] = (unsigned char)(CLIPVALUE(RGB2U(r, g, b), 0, 255));
    dp_v_data[num + offset] = (unsigned char)(CLIPVALUE(RGB2V(r, g, b), 0, 255));
  }
}


__global__ static void __RgbToNv12(const unsigned char* dpRgbData, size_t rgbPitch, unsigned char* dpNv12Data, size_t nv12Pitch, int width, int height)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int w = index % nv12Pitch;
  int h = index / nv12Pitch;

  if (w >= width || h >= height)
    return;

  unsigned char* dp_y_data = dpNv12Data;
  unsigned char* dp_u_data = dp_y_data + height * nv12Pitch;

  unsigned char r = dpRgbData[h * rgbPitch + w * 3 + 0];
  unsigned char g = dpRgbData[h * rgbPitch + w * 3 + 1];
  unsigned char b = dpRgbData[h * rgbPitch + w * 3 + 2];

  dp_y_data[h * nv12Pitch + w] = (unsigned char)CLIPVALUE(RGB2Y(r, g, b), 0, 255);
  int num = (h / 2 * width / 2 + w / 2);
  int offset = (num * 2 + 1) / width * (nv12Pitch - width);

  if (h % 2 == 0 && w % 2 == 0)
  {
    dp_u_data[num * 2 + 0 + offset] = (unsigned char)(CLIPVALUE(RGB2U(r, g, b), 0, 255));
    dp_u_data[num * 2 + 1 + offset] = (unsigned char)(CLIPVALUE(RGB2V(r, g, b), 0, 255));
  }
}

__global__ static void __RgbToYuv422p(const unsigned char* dpRgbData, size_t rgbPitch, unsigned char* dpYuv422pData, size_t yuv422pPitch, int width, int height)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int w = index % yuv422pPitch;
  int h = index / yuv422pPitch;

  if (w >= width || h >= height)
    return;

  unsigned char* dp_y_data = dpYuv422pData;
  unsigned char* dp_u_data = dp_y_data + height * yuv422pPitch;
  unsigned char* dp_v_data = dp_u_data + height / 2 * yuv422pPitch;

  unsigned char r = dpRgbData[h * rgbPitch + w * 3 + 0];
  unsigned char g = dpRgbData[h * rgbPitch + w * 3 + 1];
  unsigned char b = dpRgbData[h * rgbPitch + w * 3 + 2];

  dp_y_data[h * yuv422pPitch + w] = (unsigned char)CLIPVALUE(RGB2Y(r, g, b), 0, 255);
  int num = h * width / 2 + w / 2;
  int offset = num / width * (yuv422pPitch - width);

  if (w % 2 == 0)
  {
    dp_u_data[num + offset] = (unsigned char)(CLIPVALUE(RGB2U(r, g, b), 0, 255));
    dp_v_data[num + offset] = (unsigned char)(CLIPVALUE(RGB2V(r, g, b), 0, 255));
  }
}
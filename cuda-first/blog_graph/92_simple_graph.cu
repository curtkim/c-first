
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <iomanip>

enum { N = 500000, NSTEP = 1000, NKERNEL = 20 };

__global__ void shortKernel(float * out_d, float * in_d){
  int idx=blockIdx.x*blockDim.x+threadIdx.x;
  if(idx<N) out_d[idx]=1.23*in_d[idx];
}

int main()
{
  hipStream_t stream;
  auto blocks = 512;
  auto threads = 512;
  if (!((hipSuccess) == (hipStreamCreate(&stream)))) {
    throw std::runtime_error("cudaStreamCreate(&stream)");
  };


  float *data_in;
  hipMalloc(&data_in, N * sizeof(float));
  float *data_out;
  hipMalloc(&data_out, N * sizeof(float));

  bool graphCreated=false;
  hipGraph_t graph;
  hipGraphExec_t instance;

  std::chrono::system_clock::time_point now = std::chrono::system_clock::now();
  for(int istep=0; istep<NSTEP; istep++){
    if(!graphCreated){
      hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
      for(int ikrnl=0; ikrnl<NKERNEL; ikrnl++){
        shortKernel<<<blocks, threads, 0, stream>>>(data_out, data_in);
      }
      hipStreamEndCapture(stream, &graph);
      hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
      graphCreated=true;
    }
    hipGraphLaunch(instance, stream);

    hipStreamSynchronize(stream);
  }


  std::chrono::duration<double> d = std::chrono::duration<double>(now.time_since_epoch());
  std::cout << std::setprecision (std::numeric_limits<double>::digits10 + 1)
            << d.count() / (NSTEP*NKERNEL) << std::endl;
}

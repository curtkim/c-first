#include "hip/hip_runtime.h"
/*
 * Copyright 2020 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
* This sample demonstrates stream ordered memory allocation on a GPU using
* hipMallocAsync and cudaMemPool family of APIs.
*
* basicStreamOrderedAllocation(): demonstrates stream ordered allocation using
* hipMallocAsync/hipFreeAsync APIs with default settings.
*
* streamOrderedAllocationPostSync(): demonstrates if there's a synchronization in between allocations,
* then setting the release threshold on the pool will make sure the synchronize will not
* free memory back to the OS.
*/

// System includes
#include <stdio.h>
#include <assert.h>
#include <climits>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include "common/helper_functions.h"
#include "common/hip/hip_runtime_api.h"

#define MAX_ITER 20

/* Add two vectors on the GPU */
__global__ void vectorAddGPU(const float *a, const float *b, float *c, int N)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;

  if (idx < N) {
    c[idx] = a[idx] + b[idx];
  }
}

int basicStreamOrderedAllocation(const int dev, const int nelem, const float *a, const float *b, float *c)
{
  float *d_a, *d_b, *d_c; // Device buffers
  float errorNorm, refNorm, ref, diff;
  size_t bytes = nelem * sizeof(float);

  hipStream_t stream;
  printf("Starting basicStreamOrderedAllocation()\n");
  checkCudaErrors(hipSetDevice(dev));
  checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  checkCudaErrors(hipMallocAsync(&d_a, bytes, stream));
  checkCudaErrors(hipMallocAsync(&d_b, bytes, stream));
  checkCudaErrors(hipMallocAsync(&d_c, bytes, stream));
  checkCudaErrors(hipMemcpyAsync(d_a, a, bytes, hipMemcpyHostToDevice, stream));
  checkCudaErrors(hipMemcpyAsync(d_b, b, bytes, hipMemcpyHostToDevice, stream));

  dim3 block(256);
  dim3 grid((unsigned int)ceil(nelem/(float)block.x));
  vectorAddGPU<<<grid, block, 0, stream>>>(d_a, d_b, d_c, nelem);

  checkCudaErrors(hipFreeAsync(d_a, stream));
  checkCudaErrors(hipFreeAsync(d_b, stream));
  checkCudaErrors(hipMemcpyAsync(c, d_c, bytes, hipMemcpyDeviceToHost, stream));
  checkCudaErrors(hipFreeAsync(d_c, stream));
  checkCudaErrors(hipStreamSynchronize(stream));

  /* Compare the results */
  printf("> Checking the results from vectorAddGPU() ...\n");
  errorNorm = 0.f;
  refNorm = 0.f;

  for (int n = 0; n < nelem; n++) {
    ref = a[n] + b[n];
    diff = c[n] - ref;
    errorNorm += diff*diff;
    refNorm += ref*ref;
  }

  errorNorm = (float)sqrt((double)errorNorm);
  refNorm = (float)sqrt((double)refNorm);
  if (errorNorm/refNorm < 1.e-6f)
    printf("basicStreamOrderedAllocation PASSED\n");

  checkCudaErrors(hipStreamDestroy(stream));

  return errorNorm/refNorm < 1.e-6f ? EXIT_SUCCESS : EXIT_FAILURE;
}

// streamOrderedAllocationPostSync():
// demonstrates If the application wants the memory to persist in the pool beyond synchronization,
// then it sets the release threshold on the pool.
// This way, when the application reaches the "steady state",
// it is no longer allocating/freeing memory from the OS.
int streamOrderedAllocationPostSync(const int dev, const int nelem, const float *a, const float *b, float *c)
{
  float *d_a, *d_b, *d_c; // Device buffers
  float errorNorm, refNorm, ref, diff;
  size_t bytes = nelem * sizeof(float);

  hipStream_t stream;
  hipMemPool_t memPool;
  hipEvent_t start, end;
  printf("Starting streamOrderedAllocationPostSync()\n");
  checkCudaErrors(hipSetDevice(dev));
  checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&end));

  checkCudaErrors(hipDeviceGetDefaultMemPool(&memPool, dev));
  uint64_t thresholdVal = ULONG_MAX;
  // set high release threshold on the default pool so that hipFreeAsync will not actually release memory to the system.
  // By default, the release threshold for a memory pool is set to zero. This implies that the CUDA driver is
  // allowed to release a memory chunk back to the system as long as it does not contain any active suballocations.
  checkCudaErrors(hipMemPoolSetAttribute(memPool, hipMemPoolAttrReleaseThreshold, (void*)&thresholdVal));

  // Record the start event
  checkCudaErrors(hipEventRecord(start, stream));
  for (int i = 0; i < MAX_ITER; i++)
  {
    checkCudaErrors(hipMallocAsync(&d_a, bytes, stream));
    checkCudaErrors(hipMallocAsync(&d_b, bytes, stream));
    checkCudaErrors(hipMallocAsync(&d_c, bytes, stream));
    checkCudaErrors(hipMemcpyAsync(d_a, a, bytes, hipMemcpyHostToDevice, stream));
    checkCudaErrors(hipMemcpyAsync(d_b, b, bytes, hipMemcpyHostToDevice, stream));

    dim3 block(256);
    dim3 grid((unsigned int)ceil(nelem/(float)block.x));
    vectorAddGPU<<<grid, block, 0, stream>>>(d_a, d_b, d_c, nelem);

    checkCudaErrors(hipFreeAsync(d_a, stream));
    checkCudaErrors(hipFreeAsync(d_b, stream));
    checkCudaErrors(hipMemcpyAsync(c, d_c, bytes, hipMemcpyDeviceToHost, stream));
    checkCudaErrors(hipFreeAsync(d_c, stream));
    checkCudaErrors(hipStreamSynchronize(stream));
  }
  checkCudaErrors(hipEventRecord(end, stream));
  // Wait for the end event to complete
  checkCudaErrors(hipEventSynchronize(end));

  float msecTotal = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, end));
  printf("Total elapsed time = %f ms over %d iterations\n", msecTotal, MAX_ITER);

  /* Compare the results */
  printf("> Checking the results from vectorAddGPU() ...\n");
  errorNorm = 0.f;
  refNorm = 0.f;

  for (int n = 0; n < nelem; n++) {
    ref = a[n] + b[n];
    diff = c[n] - ref;
    errorNorm += diff*diff;
    refNorm += ref*ref;
  }

  errorNorm = (float)sqrt((double)errorNorm);
  refNorm = (float)sqrt((double)refNorm);
  if (errorNorm/refNorm < 1.e-6f)
    printf("streamOrderedAllocationPostSync PASSED\n");

  checkCudaErrors(hipStreamDestroy(stream));

  return errorNorm/refNorm < 1.e-6f ? EXIT_SUCCESS : EXIT_FAILURE;
}


int main(int argc, char **argv)
{
  int nelem;
  int dev = 0; // use default device 0
  size_t bytes;
  float *a, *b, *c; // Host

  if (checkCmdLineFlag(argc, (const char **)argv, "help"))
  {
    printf("Usage:  streamOrderedAllocation [OPTION]\n\n");
    printf("Options:\n");
    printf("  --device=[device #]  Specify the device to be used\n");
    return EXIT_SUCCESS;
  }

  dev = findCudaDevice(argc, (const char **)argv);

  int isMemPoolSupported = 0;
  checkCudaErrors(hipDeviceGetAttribute(&isMemPoolSupported, hipDeviceAttributeMemoryPoolsSupported, dev));
  if (!isMemPoolSupported)
  {
    printf("Waiving execution as device does not support Memory Pools\n");
    exit(EXIT_WAIVED);
  }

  // Allocate CPU memory.
  nelem = 1048576;
  bytes = nelem*sizeof(float);

  a = (float*) malloc(bytes);
  b = (float*) malloc(bytes);
  c = (float*) malloc(bytes);
  /* Initialize the vectors. */
  for (int n = 0; n < nelem; n++)
  {
    a[n] = rand() / (float)RAND_MAX;
    b[n] = rand() / (float)RAND_MAX;
  }

  int ret1 = basicStreamOrderedAllocation(dev, nelem, a, b, c);
  int ret2 = streamOrderedAllocationPostSync(dev, nelem, a, b, c);

  /* Memory clean up */
  free(a);
  free(b);
  free(c);

  return ((ret1 == EXIT_SUCCESS && ret2 == EXIT_SUCCESS) ? EXIT_SUCCESS : EXIT_FAILURE);
}
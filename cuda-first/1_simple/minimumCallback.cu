#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime_api.h"
#include <thread>

#define N (1024*1024)
#define M (1000000)

__global__ void cudakernel(float *buf)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    buf[i] = 1.0f * i / N;
    for(int j = 0; j < M; j++)
        buf[i] = buf[i] * buf[i] - 0.25f;
}

void CUDART_CB myStreamCallback(hipStream_t stream, hipError_t status, void *data)
{
    // Check status of GPU after stream operations are done
    checkCudaErrors(status);
    float* fData = (float*)data;
    printf("callback thread(%u) data[0] = %f\n", std::this_thread::get_id(), fData[0]);
}

int main()
{
    float data[N];
    int count = 0;
    float *d_data;

    printf("main thread(%u)\n", std::this_thread::get_id());

    hipStream_t stream;
    checkCudaErrors(hipStreamCreate(&stream));

    hipMallocAsync(&d_data, N * sizeof(float), stream);
    cudakernel<<<N/256, 256, 0, stream>>>(d_data);
    hipMemcpyAsync(data, d_data, N * sizeof(float), hipMemcpyDeviceToHost, stream);
    hipFreeAsync(d_data, stream);
    checkCudaErrors(hipStreamAddCallback(stream, myStreamCallback, data, 0));

    printf("before destroy stream\n");
    hipStreamDestroy(stream);
    printf("after destroy stream\n");

    getchar();
}